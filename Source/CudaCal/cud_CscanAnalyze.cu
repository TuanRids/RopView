#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void addKernel(int* c, const int* a, const int* b) {
    size_t i = threadIdx.x;  
    c[i] = a[i] + b[i];
}


void addWithCuda(int* c, const int* a, const int* b, size_t size) {
    int* dev_a = 0;
    int* dev_b = 0;
    int* dev_c = 0;

    hipMalloc((void**)&dev_c, size * sizeof(int));
    hipMalloc((void**)&dev_a, size * sizeof(int));
    hipMalloc((void**)&dev_b, size * sizeof(int));

    hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    addKernel << < 1, static_cast<unsigned int>(size) >> > (dev_c, dev_a, dev_b);

    hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
}
